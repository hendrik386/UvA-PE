#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <iostream>

#include <getopt.h>
#include <stdint.h>

#include "timer.h"

using namespace std;

#define THREAD_BLOCK_SIZE 512

/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}


__global__ void histogramKernel(const unsigned char* __restrict__ image, const uint64_t img_size, unsigned int* __restrict__ histogram) {

    const  uint64_t img_index = blockDim.x * blockIdx.x + threadIdx.x;
    __shared__ uint32_t block_hist[256];
    uint32_t thread_block_index = threadIdx.x;

    if (thread_block_index < 256) {
        block_hist[thread_block_index] = 0;
    }
    __syncthreads();

    if(img_index < img_size){
        atomicAdd(&block_hist[image[img_index]], 1);
    }

    __syncthreads();
    if (thread_block_index < 256) {
        uint32_t hist_block_index = blockIdx.x * 256 + threadIdx.x;
        histogram[hist_block_index] = block_hist[thread_block_index];
    }
}

__global__ void mergeKernel(uint32_t * __restrict__ block_histo, const uint32_t amout_of_blocks, uint32_t * __restrict__ histo) {
    uint32_t sum = 0;
    for (uint32_t idx = 0; idx < amout_of_blocks; ++idx) {
        uint32_t index = threadIdx.x + idx * 256;
        sum += block_histo[index];
    }

    histo[threadIdx.x] = sum;
}

void histogramCuda(const unsigned char* image, const uint64_t img_size, unsigned int* histogram, const int hist_size) {
    uint32_t blocks = img_size / THREAD_BLOCK_SIZE;
    uint32_t remainding = img_size % THREAD_BLOCK_SIZE;

    if(remainding) {
        ++blocks;
    }

    // allocate the vectors on the GPU
    unsigned char* deviceImage = NULL;
    checkCudaCall(hipMalloc((void **) &deviceImage, img_size * sizeof(unsigned char)));
    if (deviceImage == NULL) {
        cout << "could not allocate memory!" << endl;
        return;
    }
    unsigned int* deviceHisto = NULL;
    checkCudaCall(hipMalloc((void **) &deviceHisto, hist_size * sizeof(unsigned int)));
    if (deviceHisto == NULL) {
        checkCudaCall(hipFree(deviceImage));
        cout << "could not allocate memory!" << endl;
        return;
    }

    uint32_t *deviceBlockHisto = NULL;
    checkCudaCall(hipMalloc((void **) &deviceBlockHisto, blocks * hist_size * sizeof(uint32_t)));
    if (deviceBlockHisto == NULL) {
        checkCudaCall(hipFree(deviceImage));
        checkCudaCall(hipFree(deviceHisto));
        cout << "could not allocate memory!" << endl;
        return;
    }

    // hipMemset(deviceHisto, 0, hist_size * sizeof(unsigned int));
    hipMemset(deviceBlockHisto, 0, blocks * hist_size * sizeof(uint32_t));

    timer kernelTime1 = timer("kernelTime1");
    timer memoryTime = timer("memoryTime");

    uint32_t local_hist[hist_size];
    memset(local_hist, 0, hist_size * sizeof(uint32_t));

    // copy the original vectors to the GPU
    memoryTime.start();
    checkCudaCall(hipMemcpy(deviceImage, image, img_size*sizeof(unsigned char), hipMemcpyHostToDevice));
    memoryTime.stop();

    // execute kernel
    kernelTime1.start();
    histogramKernel<<<blocks, THREAD_BLOCK_SIZE>>>(deviceImage, img_size, deviceBlockHisto);
    hipDeviceSynchronize();
    checkCudaCall(hipGetLastError());

    mergeKernel<<<1, hist_size>>>(deviceBlockHisto, blocks, deviceHisto);
    hipDeviceSynchronize();

    kernelTime1.stop();

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    // copy result back
    memoryTime.start();
    checkCudaCall(hipMemcpy(histogram, deviceHisto, hist_size * sizeof(unsigned int), hipMemcpyDeviceToHost));
    memoryTime.stop();

    checkCudaCall(hipFree(deviceImage));
    checkCudaCall(hipFree(deviceHisto));
    checkCudaCall(hipFree(deviceBlockHisto));

    cout << "histogram (kernel): \t\t" << kernelTime1 << endl;
    cout << "histogram (memory): \t\t" << memoryTime << endl;
}

void histogramSeq(unsigned char* image, long img_size, unsigned int* histogram, int hist_size) {
  int i; 

  timer sequentialTime = timer("Sequential");
  
  for (i=0; i<hist_size; i++) histogram[i]=0;

  sequentialTime.start();
  for (i=0; i<img_size; i++) {
	histogram[image[i]]++;
  }
  sequentialTime.stop();
  
  cout << "histogram (sequential): \t\t" << sequentialTime << endl;

}

int main(int argc, char* argv[]) {
    uint64_t img_size = 655360;
    uint64_t hist_size = 256;

    int32_t c;

    char *file_name = NULL;
    
    while((c = getopt(argc, argv, "s:f:")) != -1) {
        switch (c)
        {    
            case 's':
                img_size = strtoul(optarg, 0, 10);
                break;
            
            case 'f':
                file_name = optarg;
                break;

            default:
                break;
        }
    }

    //Need to use __constant__ for image;

    uint8_t *image = NULL;
    unsigned int *histogramS = (unsigned int *)malloc(hist_size * sizeof(unsigned int));     
    unsigned int *histogram = (unsigned int *)malloc(hist_size * sizeof(unsigned int));

    memset(histogramS, 0, hist_size * sizeof(unsigned int));
    memset(histogram, 0, hist_size * sizeof(unsigned int));

    if (file_name != NULL){
        FILE *f;
        if (!(f = fopen(file_name, "r"))) {
            printf("unalbe to open %s", file_name);
            exit(1);
        }

        char format[2];
        unsigned imgw, imgh, maxv;

        if (fscanf(f, "%2s", format) != 1) exit(1); 
        if (format[0] != 'P' || format[1] != '2') {
            printf("only ASCII PGM input is supported");
            exit(1);
        }

        if (fscanf(f, "%u", &imgw) != 1 ||
        fscanf(f, "%u", &imgh) != 1 ||
        fscanf(f, "%u", &maxv) != 1) exit(1);
    
        img_size = imgw * imgh;
        image = (uint8_t *)malloc(img_size * sizeof(uint8_t));

        uint64_t idx;
        for (idx = 0; idx < img_size; ++idx) {
            if (fscanf(f, "%hhu", &image[idx]) != 1) {
                printf("invalid data\n");
                exit(1);
            }
        } 
    }
    else 
    {
        image = (uint8_t *)malloc(img_size * sizeof(uint8_t));
        for(long i=0; i<img_size; i++) {
            image[i] = (uint8_t) (i % hist_size);
        }
    }

    cout << "Compute the histogram of a gray image with " << img_size << " pixels." << endl;

    histogramSeq(image, img_size, histogramS, hist_size);
    histogramCuda(image, img_size, histogram, hist_size);
    
    // verify the resuls
    for(int i=0; i<hist_size; i++) {
	  if (histogram[i]!=histogramS[i]) {
            cout << "error in results! Bin " << i << " is "<< histogram[i] << ", but should be " << histogramS[i] << endl; 
            free(image);
            free(histogram);
            free(histogramS); 
            exit(1);
        }
    }
    cout << "results OK!" << endl;
     
    free(image);
    free(histogram);
    free(histogramS);         
    
    return 0;
}
