#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include "mmio.h"

#define N  512
#define M  512
#define P  512

#define REP 50

//region CPU

void matrix_mult(int left_rows, int shared_dimensions, int right_columns, float* left, float* right, float* result) {
	int row;
	int column;
	int cell;

	for(row = 0; row < left_rows; row++) {
		for(column = 0; column < right_columns; column++) {
			result[row * right_columns + column] = 0;

			for(cell = 0; cell < shared_dimensions; cell++) {
				result[row * right_columns + column] += left[row * shared_dimensions + cell] * right[cell * right_columns + column];
			}
		}
	}
}

void generate_mat(int left_rows, int shared_dimensions, int right_columns, float* left, float* right) {
	int cell;

	for(cell = 0; cell < (left_rows * shared_dimensions); cell++) {
		left[cell] = 1;
	} //cell/10;

	for(cell = 0; cell < (shared_dimensions * right_columns); cell++) {
		right[cell] = 1; //cell/5;
	}
}

void read_sparse(FILE* file, int rows, int columns, int nz, float* matrix) {
	int i;
	int row;
	int column;
	float value;

	/* NOTE: when reading in doubles, ANSI C requires the use of the "l"  */
	/*   specifier as in "%lg", "%lf", "%le", otherwise errors will occur */
	/*  (ANSI C X3.159-1989, Sec. 4.9.6.2, p. 136 lines 13-15)            */

	for(i = 0; i < nz; i++) {
		fscanf(file, "%d %d %f\n", &row, &column, &value);

		matrix[(row - 1) * columns + column - 1] = value;   /* adjust from 1-based to 0-based */
	}

}

void write_sparse(FILE* file, int rows, int columns, const float* matrix) {
	int i;
	int nz = 0;
	MM_typecode matrix_banner;

	for(i = 0; i < rows * columns; i++) {
		if(matrix[i] != 0.0) {
			nz++;
		}
	}

	mm_initialize_typecode(&matrix_banner);
	mm_set_matrix(&matrix_banner);
	mm_set_coordinate(&matrix_banner);
	mm_set_real(&matrix_banner);

	mm_write_banner(file, matrix_banner);
	mm_write_mtx_crd_size(file, rows, columns, nz);

	for(i = 0; i < rows * columns; i++) {
		if(matrix[i] != 0.0) {
			fprintf(file, "%d %d %f\n", i / columns + 1, i % columns + 1, matrix[i]);
		}
	}

}

void read_dense(FILE* matrix_file, int rows, int columns, float* matrix) {
	int row;
	int column;

	for(row = 0; row < rows; row++) {
		for(column = 0; column < columns; column++) {
			fscanf(matrix_file, "%f ", &matrix[row * columns + column]);

			//	printf("%20.19f \columns", matrix[row*(*columns)+column]);
		}
	}
}

int read_mat(int* left_rows, int* shared_dimensions, int* right_columns, int* nzA, int* nzB, FILE* left_file, FILE* right_file) {
	MM_typecode left_banner;
	MM_typecode right_banner;
	int return_code;
	int n1;

	if(mm_read_banner(left_file, &left_banner) != 0) {
		printf("Could not process Matrix Market banner for A.\n");

		return -3;
	}
	if(mm_read_banner(right_file, &right_banner) != 0) {
		printf("Could not process Matrix Market banner for B.\n");

		return -4;
	}

	if(mm_is_complex(left_banner)) {
		return -6;
	}
	if(mm_is_complex(right_banner)) {
		return -7;
	}

	if(mm_is_matrix(left_banner) && mm_is_sparse(left_banner)) {
		if((return_code = mm_read_mtx_crd_size(left_file, left_rows, shared_dimensions, nzA)) != 0) {
			return -10;
		}
	} else if(mm_is_matrix(left_banner) && mm_is_array(left_banner)) {
		*nzA = 0;
		if((return_code = mm_read_mtx_array_size(left_file, left_rows, shared_dimensions)) != 0) {
			return -11;
		}

	} else {
		return -8;
	}

	if(mm_is_matrix(right_banner) && mm_is_sparse(right_banner)) {
		if((return_code = mm_read_mtx_crd_size(right_file, &n1, right_columns, nzB)) != 0) {
			return -10;
		}
	} else if(mm_is_matrix(right_banner) && mm_is_array(right_banner)) {
		*nzB = 0;
		if((return_code = mm_read_mtx_array_size(right_file, &n1, right_columns)) != 0) {
			return -11;
		}

	} else {
		return -9;
	}

	if(*shared_dimensions != n1) {
		return -15;
	}

	return 0;
	/* find out size of sparse matrix .... */
}

//endregion

//region GPU

float* device_left;

float* device_right;

float* device_result;

__global__ void gpu_matrix_mult(int left_rows, int shared_dimensions, int right_columns, float* left, float* right, float* result) {
	// Get position in the result matrix
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int column = blockIdx.x * blockDim.x + threadIdx.x;

	// Check if we're inside the bounds of the result matrix
	if(row < left_rows && column < right_columns) {
		// Convert coordinates to 1D index
		int index = row * right_columns + column;

		// Initialize result value
		result[index] = 0;

		// Perform multiplication operation
		int cell;
		for(cell = 0; cell < shared_dimensions; cell++) {
			result[index] += left[row * shared_dimensions + cell] * right[cell * right_columns + column];
		}
	}
}

void output_gpu_info() {
	int runtimeVersion = 0;
	int driverVersion = 0;
	hipRuntimeGetVersion(&runtimeVersion);
	hipDriverGetVersion(&driverVersion);

	printf("CUDA runtime version: %d\n", runtimeVersion);
	printf("CUDA driver version: %d\n", driverVersion);
}

static void checkCudaCall(hipError_t result) {
	if(result != hipSuccess) {
		printf("CUDA error %s: %s\n", hipGetErrorName(result), hipGetErrorString(result));

		exit(1);
	}
}

//endregion

int main(int argument_count, char** argument_values) {
	#ifdef TIMING
	struct timeval before;
	struct timeval after;
	#endif

	int nzA = 0;
	int nzB = 0;
	FILE* left_file;
	FILE* right_file;

	int left_rows;
	int shared_dimensions;
	int right_columns;
	#ifdef GENERATE
	left_rows = M;
	shared_dimensions = N;
	right_columns = P;
	#else
	if(argument_count < 3) {
		fprintf(stderr, "Usage: %s [martix1] [matrix2] [resultmatrix] \n", argument_values[0]);
		exit(1);
	} else {
		if((left_file = fopen(argument_values[1], "rt")) == NULL) {
			exit(1);
		}

		if((right_file = fopen(argument_values[2], "rt")) == NULL) {
			exit(2);
		}

		int error_code = read_mat(&left_rows, &shared_dimensions, &right_columns, &nzA, &nzB, left_file, right_file);
		if(error_code == -15) {
			printf("Matrices are incompatible! \n");
			fclose(left_file);
			fclose(right_file);
			exit(1);
		}
	}
	#endif

	float* left = (float*) calloc(left_rows * shared_dimensions, sizeof(float));
	if(left == NULL) {
		printf("Out of memory left! \n");
		exit(1);
	}

	float* right = (float*) calloc(shared_dimensions * right_columns, sizeof(float));
	if(right == NULL) {
		printf("Out of memory right! \n");
		exit(1);
	}

	#ifdef GENERATE
	generate_mat(left_rows, shared_dimensions, right_columns, left, right);
	#else
	if(nzA > 0) {
		read_sparse(left_file, left_rows, shared_dimensions, nzA, left);
	} else {
		read_dense(left_file, left_rows, shared_dimensions, left);
	}

	if(nzB > 0) {
		read_sparse(right_file, shared_dimensions, right_columns, nzB, right);
	} else {
		read_dense(right_file, shared_dimensions, right_columns, right);
	}

	fclose(left_file);
	fclose(right_file);
	#endif

	float* result = (float*) calloc(left_rows * right_columns, sizeof(float));
	if(result == NULL) {
		printf("Out of memory C1! \n");
		exit(1);
	}
	// C2 = (float *)calloc(N*P,sizeof(float));
	// if (C2==NULL) {printf("Out of memory C2! \n"); exit(1);}

	// Initialize cuda
	output_gpu_info();
	dim3 block_size(16, 16);
	dim3 grid_size((right_columns + block_size.x - 1) / block_size.x, (left_rows + block_size.y - 1) / block_size.y);
	printf("Grid size: (%d, %d, %d)\n", grid_size.x, grid_size.y, grid_size.z);
	printf("Block size: (%d, %d, %d)\n", block_size.x, block_size.y, block_size.z);
	hipMalloc((void**) &device_left, left_rows * shared_dimensions * sizeof(float));
	hipMalloc((void**) &device_right, shared_dimensions * right_columns * sizeof(float));
	hipMalloc((void**) &device_result, left_rows * right_columns * sizeof(float));
	hipMemcpy(device_left, left, left_rows * shared_dimensions * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(device_right, right, shared_dimensions * right_columns * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(device_result, result, left_rows * right_columns * sizeof(float), hipMemcpyHostToDevice);

	// Naive implementation
	#ifdef TIMING
	gettimeofday(&before, NULL);
	#endif

	int r;
	for(r = 0; r < REP; r++) {
		gpu_matrix_mult << < grid_size, block_size >> > (left_rows, shared_dimensions, right_columns, device_left, device_right, device_result);
	}
	hipDeviceSynchronize();
	checkCudaCall(hipGetLastError());

	#ifdef TIMING
	gettimeofday(&after, NULL);
	printf("Reference code: %10.6f seconds \n", ((after.tv_sec + (after.tv_usec / 1000000.0)) - (before.tv_sec + (before.tv_usec / 1000000.0))) / REP);
	#endif

	// Retrieve GPU data and clean up
	hipMemcpy(left, device_left, left_rows * shared_dimensions * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(right, device_right, shared_dimensions * right_columns * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(result, device_result, left_rows * right_columns * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(device_left);
	hipFree(device_right);
	hipFree(device_result);

	FILE* result_file;
	#ifdef GENERATE
	if ((result_file = fopen("gen_result.mtx", "wt")) == NULL) {
		exit(3);
	}
	#else
	if((result_file = fopen(argument_values[3], "wt")) == NULL) {
		exit(3);
	}
	#endif
	write_sparse(result_file, left_rows, right_columns, result);
	fclose(result_file);

	free(left);
	free(right);
	free(result);
	// free(C2);
}

